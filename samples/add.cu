#include <hip/hip_runtime.h>
#include <stdio.h>

// CPU addition
void add(float* A_h, float* B_h, float* C_h, int n) {
    for (int i = 0; i < n; i++) C_h[i] = A_h[i] + B_h[i];
}

// GPU addition
__global__ void addKernel(float* A, float* B, float* C, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) C[i] = A[i] + B[i];
}

void vecAdd(float* A, float* B, float* C, int n) {
    // Dados do programa
    int size = n * sizeof(float);
    float *d_A, *d_B, *d_C;

    //  01 - Requisita memoria ao device para executar
    // bloco de código com os valores A,B,C
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // 02 - Chama bloco que executará operações no device
    addKernel <<<ceil(n/256.0), 256>>>(d_A, d_B, d_C, n);

    // 03 - Copia resultado do device para o host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Libera a memoria alocada
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}