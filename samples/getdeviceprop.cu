
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char **argv) {
    int devCount;
    hipGetDeviceCount(&devCount);
    printf("Number of device: %d", devCount);

    hipDeviceProp_t devprop;
    hipGetDeviceProperties(&deviceProp, 0);
    printf("%s using %d: %s\n", argv[0], 0, deviceProp.name);

}